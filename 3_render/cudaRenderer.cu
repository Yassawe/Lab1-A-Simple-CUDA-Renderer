#include "hip/hip_runtime.h"
#include <string>
#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaRenderer.h"
#include "image.h"
#include "noise.h"
#include "sceneLoader.h"
#include "util.h"


#define cudaCheckError(ans) { cudaAssert((ans), __FILE__ , __LINE__ ); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


////////////////////////////////////////////////////////////////////////////////////////
// Putting all the cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

struct GlobalConstants {

    SceneName sceneName;

    int numCircles;
    float* position;
    float* velocity;
    float* color;
    float* radius;

    int imageWidth;
    int imageHeight;
    float* imageData;
};

// Global variable that is in scope, but read-only, for all cuda
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (we didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants cuConstRendererParams;

// read-only lookup tables used to quickly compute noise (needed by
// advanceAnimation for the snowflake scene)
__constant__ int    cuConstNoiseYPermutationTable[256];
__constant__ int    cuConstNoiseXPermutationTable[256];
__constant__ float  cuConstNoise1DValueTable[256];

// color ramp table needed for the color ramp lookup shader
#define COLOR_MAP_SIZE 5
__constant__ float  cuConstColorRamp[COLOR_MAP_SIZE][3];


// including parts of the CUDA code from external files to keep this
// file simpler and to seperate code that should not be modified
#include "noiseCuda.cu_inl"
#include "lookupColor.cu_inl"


// kernelClearImageSnowflake -- (CUDA device code)
//
// Clear the image, setting the image to the white-gray gradation that
// is used in the snowflake image
__global__ void kernelClearImageSnowflake() {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float shade = .4f + .45f * static_cast<float>(height-imageY) / height;
    float4 value = make_float4(shade, shade, shade, 1.f);

    // write to global memory: As an optimization, I use a float4
    // store, that results in more efficient code than if I coded this
    // up as four seperate fp32 stores.
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelClearImage --  (CUDA device code)
//
// Clear the image, setting all pixels to the specified color rgba
__global__ void kernelClearImage(float r, float g, float b, float a) {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float4 value = make_float4(r, g, b, a);

    // write to global memory: As an optimization, I use a float4
    // store, that results in more efficient code than if I coded this
    // up as four seperate fp32 stores.
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelAdvanceFireWorks
//
// Update the position of the fireworks (if circle is firework)
__global__ void kernelAdvanceFireWorks() {
    const float dt = 1.f / 60.f;
    const float pi = 3.14159;
    const float maxDist = 0.25f;

    float* velocity = cuConstRendererParams.velocity;
    float* position = cuConstRendererParams.position;
    float* radius = cuConstRendererParams.radius;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numCircles)
        return;

    if (0 <= index && index < NUM_FIREWORKS) { // firework center; no update
        return;
    }

    // determine the fire-work center/spark indices
    int fIdx = (index - NUM_FIREWORKS) / NUM_SPARKS;
    int sfIdx = (index - NUM_FIREWORKS) % NUM_SPARKS;

    int index3i = 3 * fIdx;
    int sIdx = NUM_FIREWORKS + fIdx * NUM_SPARKS + sfIdx;
    int index3j = 3 * sIdx;

    float cx = position[index3i];
    float cy = position[index3i+1];

    // update position
    position[index3j] += velocity[index3j] * dt;
    position[index3j+1] += velocity[index3j+1] * dt;

    // fire-work sparks
    float sx = position[index3j];
    float sy = position[index3j+1];

    // compute vector from firework-spark
    float cxsx = sx - cx;
    float cysy = sy - cy;

    // compute distance from fire-work
    float dist = sqrt(cxsx * cxsx + cysy * cysy);
    if (dist > maxDist) { // restore to starting position
        // random starting position on fire-work's rim
        float angle = (sfIdx * 2 * pi)/NUM_SPARKS;
        float sinA = sin(angle);
        float cosA = cos(angle);
        float x = cosA * radius[fIdx];
        float y = sinA * radius[fIdx];

        position[index3j] = position[index3i] + x;
        position[index3j+1] = position[index3i+1] + y;
        position[index3j+2] = 0.0f;

        // travel scaled unit length
        velocity[index3j] = cosA/5.0;
        velocity[index3j+1] = sinA/5.0;
        velocity[index3j+2] = 0.0f;
    }
}

// kernelAdvanceHypnosis
//
// Update the radius/color of the circles
__global__ void kernelAdvanceHypnosis() {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numCircles)
        return;

    float* radius = cuConstRendererParams.radius;

    float cutOff = 0.5f;
    // place circle back in center after reaching threshold radisus
    if (radius[index] > cutOff) {
        radius[index] = 0.02f;
    } else {
        radius[index] += 0.01f;
    }
}


// kernelAdvanceBouncingBalls
//
// Update the positino of the balls
__global__ void kernelAdvanceBouncingBalls() {
    const float dt = 1.f / 60.f;
    const float kGravity = -2.8f; // sorry Newton
    const float kDragCoeff = -0.8f;
    const float epsilon = 0.001f;

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numCircles)
        return;

    float* velocity = cuConstRendererParams.velocity;
    float* position = cuConstRendererParams.position;

    int index3 = 3 * index;
    // reverse velocity if center position < 0
    float oldVelocity = velocity[index3+1];
    float oldPosition = position[index3+1];

    if (oldVelocity == 0.f && oldPosition == 0.f) { // stop-condition
        return;
    }

    if (position[index3+1] < 0 && oldVelocity < 0.f) { // bounce ball
        velocity[index3+1] *= kDragCoeff;
    }

    // update velocity: v = u + at (only along y-axis)
    velocity[index3+1] += kGravity * dt;

    // update positions (only along y-axis)
    position[index3+1] += velocity[index3+1] * dt;

    if (fabsf(velocity[index3+1] - oldVelocity) < epsilon
        && oldPosition < 0.0f
        && fabsf(position[index3+1]-oldPosition) < epsilon) { // stop ball
        velocity[index3+1] = 0.f;
        position[index3+1] = 0.f;
    }
}

// kernelAdvanceSnowflake -- (CUDA device code)
//
// move the snowflake animation forward one time step.  Updates circle
// positions and velocities.  Note how the position of the snowflake
// is reset if it moves off the left, right, or bottom of the screen.
__global__ void kernelAdvanceSnowflake() {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numCircles)
        return;

    const float dt = 1.f / 60.f;
    const float kGravity = -1.8f; // sorry Newton
    const float kDragCoeff = 2.f;

    int index3 = 3 * index;

    float* positionPtr = &cuConstRendererParams.position[index3];
    float* velocityPtr = &cuConstRendererParams.velocity[index3];

    // loads from global memory
    float3 position = *((float3*)positionPtr);
    float3 velocity = *((float3*)velocityPtr);

    // hack to make farther circles move more slowly, giving the
    // illusion of parallax
    float forceScaling = fmin(fmax(1.f - position.z, .1f), 1.f); // clamp

    // add some noise to the motion to make the snow flutter
    float3 noiseInput;
    noiseInput.x = 10.f * position.x;
    noiseInput.y = 10.f * position.y;
    noiseInput.z = 255.f * position.z;
    float2 noiseForce = cudaVec2CellNoise(noiseInput, index);
    noiseForce.x *= 7.5f;
    noiseForce.y *= 5.f;

    // drag
    float2 dragForce;
    dragForce.x = -1.f * kDragCoeff * velocity.x;
    dragForce.y = -1.f * kDragCoeff * velocity.y;

    // update positions
    position.x += velocity.x * dt;
    position.y += velocity.y * dt;

    // update velocities
    velocity.x += forceScaling * (noiseForce.x + dragForce.y) * dt;
    velocity.y += forceScaling * (kGravity + noiseForce.y + dragForce.y) * dt;

    float radius = cuConstRendererParams.radius[index];

    // if the snowflake has moved off the left, right or bottom of
    // the screen, place it back at the top and give it a
    // pseudorandom x position and velocity.
    if ( (position.y + radius < 0.f) ||
         (position.x + radius) < -0.f ||
         (position.x - radius) > 1.f)
    {
        noiseInput.x = 255.f * position.x;
        noiseInput.y = 255.f * position.y;
        noiseInput.z = 255.f * position.z;
        noiseForce = cudaVec2CellNoise(noiseInput, index);

        position.x = .5f + .5f * noiseForce.x;
        position.y = 1.35f + radius;

        // restart from 0 vertical velocity.  Choose a
        // pseudo-random horizontal velocity.
        velocity.x = 2.f * noiseForce.y;
        velocity.y = 0.f;
    }

    // store updated positions and velocities to global memory
    *((float3*)positionPtr) = position;
    *((float3*)velocityPtr) = velocity;
}

// shadePixel -- (CUDA device code)
//
// given a pixel and a circle, determines the contribution to the
// pixel from the circle.  Update of the image is done in this
// function.  Called by kernelRenderCircles()
__device__ __inline__ void
shadePixel(int circleIndex, float2 pixelCenter, float3 p, float4* imagePtr) {

    float diffX = p.x - pixelCenter.x;
    float diffY = p.y - pixelCenter.y;
    float pixelDist = diffX * diffX + diffY * diffY;

    float rad = cuConstRendererParams.radius[circleIndex];;
    float maxDist = rad * rad;

    // circle does not contribute to the image
    if (pixelDist > maxDist)
        return;

    float3 rgb;
    float alpha;

    // there is a non-zero contribution.  Now compute the shading value

    // suggestion: This conditional is in the inner loop.  Although it
    // will evaluate the same for all threads, there is overhead in
    // setting up the lane masks etc to implement the conditional.  It
    // would be wise to perform this logic outside of the loop next in
    // kernelRenderCircles.  (If feeling good about yourself, you
    // could use some specialized template magic).
    if (cuConstRendererParams.sceneName == SNOWFLAKES || cuConstRendererParams.sceneName == SNOWFLAKES_SINGLE_FRAME) {

        const float kCircleMaxAlpha = .5f;
        const float falloffScale = 4.f;

        float normPixelDist = sqrt(pixelDist) / rad;
        rgb = lookupColor(normPixelDist);

        float maxAlpha = .6f + .4f * (1.f-p.z);
        maxAlpha = kCircleMaxAlpha * fmaxf(fminf(maxAlpha, 1.f), 0.f); // kCircleMaxAlpha * clamped value
        alpha = maxAlpha * exp(-1.f * falloffScale * normPixelDist * normPixelDist);

    } else {
        // simple: each circle has an assigned color
        int index3 = 3 * circleIndex;
        rgb = *(float3*)&(cuConstRendererParams.color[index3]);
        alpha = .5f;
    }

    float oneMinusAlpha = 1.f - alpha;

    // BEGIN SHOULD-BE-ATOMIC REGION
    // global memory read

    float4 existingColor = *imagePtr;
    float4 newColor;
    newColor.x = alpha * rgb.x + oneMinusAlpha * existingColor.x;
    newColor.y = alpha * rgb.y + oneMinusAlpha * existingColor.y;
    newColor.z = alpha * rgb.z + oneMinusAlpha * existingColor.z;
    newColor.w = alpha + existingColor.w;

    // global memory write
    *imagePtr = newColor;

    // END SHOULD-BE-ATOMIC REGION
}

// kernelRenderCircles -- (CUDA device code)
//
// Each thread renders a circle.  Since there is no protection to
// ensure order of update or mutual exclusion on the output image, the
// resulting image will be incorrect.
__global__ void kernelRenderCircles() {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numCircles)
        return;

    int index3 = 3 * index;

    // read position and radius
    float3 p = *(float3*)(&cuConstRendererParams.position[index3]);
    float  rad = cuConstRendererParams.radius[index];

    // compute the bounding box of the circle. The bound is in integer
    // screen coordinates, so it's clamped to the edges of the screen.
    short imageWidth = cuConstRendererParams.imageWidth;
    short imageHeight = cuConstRendererParams.imageHeight;
    short minX = static_cast<short>(imageWidth * (p.x - rad));
    short maxX = static_cast<short>(imageWidth * (p.x + rad)) + 1;
    short minY = static_cast<short>(imageHeight * (p.y - rad));
    short maxY = static_cast<short>(imageHeight * (p.y + rad)) + 1;

    // a bunch of clamps.  Is there a CUDA built-in for this?
    short screenMinX = (minX > 0) ? ((minX < imageWidth) ? minX : imageWidth) : 0;
    short screenMaxX = (maxX > 0) ? ((maxX < imageWidth) ? maxX : imageWidth) : 0;
    short screenMinY = (minY > 0) ? ((minY < imageHeight) ? minY : imageHeight) : 0;
    short screenMaxY = (maxY > 0) ? ((maxY < imageHeight) ? maxY : imageHeight) : 0;

    float invWidth = 1.f / imageWidth;
    float invHeight = 1.f / imageHeight;

    // for all pixels in the bonding box
    for (int pixelY=screenMinY; pixelY<screenMaxY; pixelY++) {
        float4* imgPtr = (float4*)(&cuConstRendererParams.imageData[4 * (pixelY * imageWidth + screenMinX)]);
        for (int pixelX=screenMinX; pixelX<screenMaxX; pixelX++) {
            float2 pixelCenterNorm = make_float2(invWidth * (static_cast<float>(pixelX) + 0.5f),
                                                 invHeight * (static_cast<float>(pixelY) + 0.5f));
            shadePixel(index, pixelCenterNorm, p, imgPtr);
            imgPtr++;
        }
    }
}
/////////////////////////////[MY CHANGES (KERNELS) START HERE]///////////////////////////////////////


#define BLOCK_SIZE 32

// naive version of pixel parallelism via image blocking (shared memory is not used)
// this approach is bad, since the majority of the threads will do nothing
__global__ void naivePixelParallelism() {
    uint bx = blockIdx.x;
    uint by = blockIdx.y;

    
    uint pixelX = bx*BLOCK_SIZE + threadIdx.x;
    uint pixelY = by*BLOCK_SIZE + threadIdx.y;

    uint imW = cuConstRendererParams.imageWidth;
    uint imH = cuConstRendererParams.imageHeight;

    if (pixelX>imW || pixelY>imH){
        return; 
    }

    float normX = 1.f/imW;
    float normY = 1.f/imH;

    float2 pixelCenterNorm = make_float2(normX * (static_cast<float>(pixelX) + 0.5f), normY * (static_cast<float>(pixelY) + 0.5f));

    float4* imgPtr = (float4*)(&cuConstRendererParams.imageData[4 * (pixelY * imW + pixelX)]); 

    for (uint i = 0; i<cuConstRendererParams.numCircles; i++){
        float3 circlePosition = *(float3*)(&cuConstRendererParams.position[3*i]);
        shadePixel(i, pixelCenterNorm, circlePosition, imgPtr);
    }

}

/////////////////////////////less naive pixel parallelism (fetch "valid" circles concurrently)///////////////////////////

#include "circleBoxTest.cu_inl"

#define SCAN_BLOCK_DIM 1024
#include "exclusiveScan.cu_inl"

//helper functions:

__device__ __inline__ void checkCircles(int index, int threadId, int totalCircles, float L, float R, float T, float B, uint* tempIdx, uint* mask, int* len){
    int globalCircleIndex = index + threadId;
    
    if (globalCircleIndex>totalCircles){
        return;
    }
    
    float rad = cuConstRendererParams.radius[globalCircleIndex];
    float3 circlePosition = *(float3*)(&cuConstRendererParams.position[3*globalCircleIndex]);

    if (circleInBox(circlePosition.x, circlePosition.y, rad, L, R, T, B)){
        tempIdx[threadId] = threadId;
        mask[threadId] = 1;
        atomicAdd(len, 1);
    }
}

__device__ __inline__ void constructValidIdx(int threadId, uint* tempIdx, uint* mask, uint* offset, uint* validIdx){
    // after exclusive prefix sum, offset contains the in-order index of the valid circle. (valid means it exists in the block)
    if(mask[threadId]==1){
        validIdx[offset[threadId]] = tempIdx[threadId]; 
    }
}


// main kernel:
__global__ void lessNaivePixelParallelism() {
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int pixelX = bx*BLOCK_SIZE + threadIdx.x;
    int pixelY = by*BLOCK_SIZE + threadIdx.y;

    int imW = cuConstRendererParams.imageWidth;
    int imH = cuConstRendererParams.imageHeight;

    float normX = 1.f/imW;
    float normY = 1.f/imH;

    float4* imgPtr = nullptr;
    if (pixelX<imW && pixelY<imH){
        imgPtr = (float4*)(&cuConstRendererParams.imageData[4 * (pixelY * imW + pixelX)]); 
    }

    float2 pixelCenterNorm = make_float2(normX * (static_cast<float>(pixelX) + 0.5f), normY * (static_cast<float>(pixelY) + 0.5f));
    

    // boundaries of current block in terms of normalized coordinates. LRBT -  left, right, bottom, top 
    float L = bx*BLOCK_SIZE*normX;
    float R = (bx*BLOCK_SIZE + BLOCK_SIZE)*normX;
    float B = by*BLOCK_SIZE*normY;
    float T = (by*BLOCK_SIZE+BLOCK_SIZE)*normY;
     
    // the idea of optimization is to iterate over several circles at once, i.e. something akin to batches, and then select only those circles that are inside the block
    // to do so, i need shared array valididx containing only indecies of circles that are present in the block.  

    int threadId = threadIdx.y * BLOCK_SIZE + threadIdx.x;

    const uint batchsize = BLOCK_SIZE*BLOCK_SIZE; //can iterate over 32*32 = 1024 circles at a time

    __shared__ uint tempIdx[batchsize];
    __shared__ uint mask[batchsize];
    __shared__ uint offset[batchsize];
    __shared__ uint validIdx[batchsize];
    __shared__ uint scratch[batchsize*2];

    __shared__ int len;

    int totalCircles = cuConstRendererParams.numCircles; 

    for (int i = 0; i<totalCircles; i+=batchsize){
        len = 0;
        tempIdx[threadId] = 0;
        mask[threadId] = 0;
        offset[threadId] = 0;
        validIdx[threadId] = 0;
        __syncthreads();


        checkCircles(i, threadId, totalCircles, L, R, T, B, tempIdx, mask, &len);
        __syncthreads();

        sharedMemExclusiveScan(threadId, mask, offset, scratch, batchsize);
        __syncthreads();

        constructValidIdx(threadId, tempIdx, mask, offset, validIdx);
        __syncthreads();
        
        if (pixelX<imW && pixelY<imH){
            for (int j = 0; j<len; j++){
                int index = i + validIdx[j];
                
                if (index>totalCircles){
                    break;
                }

                float3 circlePosition = *(float3*)(&cuConstRendererParams.position[3*index]);
                shadePixel(index, pixelCenterNorm, circlePosition, imgPtr);
            }
        }

        __syncthreads();
        
    }
}

/////////////////////////////////////[really wacky optimization, basically double everything]///////////////////////////////////////////////////

//optimal scan implementations trade shared memory for time
//i trade time for shared memory, to process 2048 circles per iteration
//because i cannot use buffer[2*batchsize] that is required for optimal scan implementatons. 

__device__ __inline__ void stupidscan(int virtualThreadId, uint* in, uint* out){ 
    uint sum = 0;

    for(int i=0; i<virtualThreadId; i++){
        sum+=in[i];
    }

    out[virtualThreadId] = sum;
    out[virtualThreadId+1] = sum + in[virtualThreadId]; //bullshit
    
    
}

__device__ __inline__ void checkCircles2(int index, int virtualThreadId, int totalCircles, float L, float R, float T, float B, uint* tempIdx, uint* mask, int* len){
    int circleindex1 = index + virtualThreadId;
    int circleindex2 = circleindex1+1;
    
    if (circleindex1>totalCircles){
        return;
    }
    
    float rad1 = cuConstRendererParams.radius[circleindex1];
    float rad2 = cuConstRendererParams.radius[circleindex2];

    float3 position1 = *(float3*)(&cuConstRendererParams.position[3*circleindex1]);
    float3 position2 = *(float3*)(&cuConstRendererParams.position[3*circleindex2]);

    if (circleInBox(position1.x, position1.y, rad1, L, R, T, B)){
        tempIdx[virtualThreadId] = virtualThreadId;
        mask[virtualThreadId] = 1;
        atomicAdd(len, 1);
    }

    if (circleInBox(position2.x, position2.y, rad2, L, R, T, B)){
        tempIdx[virtualThreadId+1] = virtualThreadId+1;
        mask[virtualThreadId+1] = 1;
        atomicAdd(len, 1);
    }
}

__device__ __inline__ void constructValidIdx2(int virtualThreadId, uint* tempIdx, uint* mask, uint* offset, uint* validIdx){
    if(mask[virtualThreadId]==1){
        validIdx[offset[virtualThreadId]] = tempIdx[virtualThreadId]; 
    }

    if(mask[virtualThreadId+1]==1){
        validIdx[offset[virtualThreadId+1]] = tempIdx[virtualThreadId+1]; 
    }

}

__global__ void doubleEverythingPixelParallel() {
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int pixelX = bx*BLOCK_SIZE + threadIdx.x;
    int pixelY = by*BLOCK_SIZE + threadIdx.y;

    int imW = cuConstRendererParams.imageWidth;
    int imH = cuConstRendererParams.imageHeight;

    float normX = 1.f/imW;
    float normY = 1.f/imH;

    float4* imgPtr = nullptr;
    if (pixelX<imW && pixelY<imH){
        imgPtr = (float4*)(&cuConstRendererParams.imageData[4 * (pixelY * imW + pixelX)]); 
    }

    float2 pixelCenterNorm = make_float2(normX * (static_cast<float>(pixelX) + 0.5f), normY * (static_cast<float>(pixelY) + 0.5f));
    

    float L = bx*BLOCK_SIZE*normX;
    float R = (bx*BLOCK_SIZE + BLOCK_SIZE)*normX;
    float B = by*BLOCK_SIZE*normY;
    float T = (by*BLOCK_SIZE+BLOCK_SIZE)*normY;
     
    const uint batchsize = 2*BLOCK_SIZE*BLOCK_SIZE; 

    __shared__ uint tempIdx[batchsize];
    __shared__ uint mask[batchsize];
    __shared__ uint offset[batchsize];
    __shared__ uint validIdx[batchsize];
    
    __shared__ int len;

    int totalCircles = cuConstRendererParams.numCircles;
    
    int threadId = threadIdx.y * BLOCK_SIZE + threadIdx.x;
    int virtualThreadId = 2*threadId;

    
    for (int i = 0; i<totalCircles; i+=batchsize){
        len = 0;

        tempIdx[virtualThreadId] = 0;
        tempIdx[virtualThreadId+1] = 0;

        mask[virtualThreadId] = 0;
        mask[virtualThreadId+1] = 0;

        offset[virtualThreadId] = 0;
        offset[virtualThreadId+1] = 0;

        validIdx[virtualThreadId] = 0;
        validIdx[virtualThreadId+1] = 0;

        __syncthreads();


        checkCircles2(i, virtualThreadId, totalCircles, L, R, T, B, tempIdx, mask, &len);
        __syncthreads();

        stupidscan(virtualThreadId, mask, offset);
        __syncthreads();

        constructValidIdx2(virtualThreadId, tempIdx, mask, offset, validIdx);
        __syncthreads();
        
        if (pixelX<imW && pixelY<imH){
            for (int j = 0; j<len; j++){
                int index = i + validIdx[j];
                
                if (index>totalCircles){
                    break;
                }

                float3 circlePosition = *(float3*)(&cuConstRendererParams.position[3*index]);
                shadePixel(index, pixelCenterNorm, circlePosition, imgPtr);
            }
        }

        __syncthreads();
        
    }
}


/////////////////////////////////////[CHANGES END HERE]///////////////////////////////////////////////////


CudaRenderer::CudaRenderer() {
    image = NULL;

    numCircles = 0;
    position = NULL;
    velocity = NULL;
    color = NULL;
    radius = NULL;

    cudaDevicePosition = NULL;
    cudaDeviceVelocity = NULL;
    cudaDeviceColor = NULL;
    cudaDeviceRadius = NULL;
    cudaDeviceImageData = NULL;
}

CudaRenderer::~CudaRenderer() {

    if (image) {
        delete image;
    }

    if (position) {
        delete [] position;
        delete [] velocity;
        delete [] color;
        delete [] radius;
    }

    if (cudaDevicePosition) {
        hipFree(cudaDevicePosition);
        hipFree(cudaDeviceVelocity);
        hipFree(cudaDeviceColor);
        hipFree(cudaDeviceRadius);
        hipFree(cudaDeviceImageData);
    }
}

const Image*
CudaRenderer::getImage() {

    // need to copy contents of the rendered image from device memory
    // before we expose the Image object to the caller

    printf("Copying image data from device\n");

    hipMemcpy(image->data,
               cudaDeviceImageData,
               sizeof(float) * 4 * image->width * image->height,
               hipMemcpyDeviceToHost);

    return image;
}

void
CudaRenderer::loadScene(SceneName scene) {
    sceneName = scene;
    loadCircleScene(sceneName, numCircles, position, velocity, color, radius);
}

void
CudaRenderer::setup() {

    int deviceCount = 0;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");

    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy

    hipMalloc(&cudaDevicePosition, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceVelocity, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceColor, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceRadius, sizeof(float) * numCircles);
    hipMalloc(&cudaDeviceImageData, sizeof(float) * 4 * image->width * image->height);

    hipMemcpy(cudaDevicePosition, position, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceVelocity, velocity, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceColor, color, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceRadius, radius, sizeof(float) * numCircles, hipMemcpyHostToDevice);

    // Initialize parameters in constant memory.  We didn't talk about
    // constant memory in class, but the use of read-only constant
    // memory here is an optimization over just sticking these values
    // in device global memory.  NVIDIA GPUs have a few special tricks
    // for optimizing access to constant memory.  Using global memory
    // here would have worked just as well.  See the Programmer's
    // Guide for more information about constant memory.

    GlobalConstants params;
    params.sceneName = sceneName;
    params.numCircles = numCircles;
    params.imageWidth = image->width;
    params.imageHeight = image->height;
    params.position = cudaDevicePosition;
    params.velocity = cudaDeviceVelocity;
    params.color = cudaDeviceColor;
    params.radius = cudaDeviceRadius;
    params.imageData = cudaDeviceImageData;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

    // also need to copy over the noise lookup tables, so we can
    // implement noise on the GPU
    int* permX;
    int* permY;
    float* value1D;
    getNoiseTables(&permX, &permY, &value1D);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseXPermutationTable), permX, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseYPermutationTable), permY, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoise1DValueTable), value1D, sizeof(float) * 256);

    // last, copy over the color table that's used by the shading
    // function for circles in the snowflake demo

    float lookupTable[COLOR_MAP_SIZE][3] = {
        {1.f, 1.f, 1.f},
        {1.f, 1.f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, 0.8f, 1.f},
    };

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstColorRamp), lookupTable, sizeof(float) * 3 * COLOR_MAP_SIZE);

}

// allocOutputImage --
//
// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void
CudaRenderer::allocOutputImage(int width, int height) {

    if (image)
        delete image;
    image = new Image(width, height);
}

// clearImage --
//
// Clear's the renderer's target image.  The state of the image after
// the clear depends on the scene being rendered.
void
CudaRenderer::clearImage() {

    // 256 threads per block is a healthy number
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(
        (image->width + blockDim.x - 1) / blockDim.x,
        (image->height + blockDim.y - 1) / blockDim.y);

    if (sceneName == SNOWFLAKES || sceneName == SNOWFLAKES_SINGLE_FRAME) {
        kernelClearImageSnowflake<<<gridDim, blockDim>>>();
    } else {
        kernelClearImage<<<gridDim, blockDim>>>(1.f, 1.f, 1.f, 1.f);
    }
    hipDeviceSynchronize();
}

// advanceAnimation --
//
// Advance the simulation one time step.  Updates all circle positions
// and velocities
void
CudaRenderer::advanceAnimation() {
     // 256 threads per block is a healthy number
    dim3 blockDim(256, 1);
    dim3 gridDim((numCircles + blockDim.x - 1) / blockDim.x);

    // only the snowflake scene has animation
    if (sceneName == SNOWFLAKES) {
        kernelAdvanceSnowflake<<<gridDim, blockDim>>>();
    } else if (sceneName == BOUNCING_BALLS) {
        kernelAdvanceBouncingBalls<<<gridDim, blockDim>>>();
    } else if (sceneName == HYPNOSIS) {
        kernelAdvanceHypnosis<<<gridDim, blockDim>>>();
    } else if (sceneName == FIREWORKS) {
        kernelAdvanceFireWorks<<<gridDim, blockDim>>>();
    }
    hipDeviceSynchronize();
}


void
CudaRenderer::render() {

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((image->width - 1)/BLOCK_SIZE+1, (image->height - 1)/BLOCK_SIZE+1);

    // if(numCircles<10){
    //     naivePixelParallelism<<<gridDim, blockDim>>>();
    // }
    // else{
    //     lessNaivePixelParallelism<<<gridDim, blockDim>>>();
    // }

    doubleEverythingPixelParallel<<<gridDim, blockDim>>>();
    
    cudaCheckError(hipPeekAtLastError());
    cudaCheckError(hipDeviceSynchronize());
    
}
